#include "hip/hip_runtime.h"
#include "kernels.cuh"

struct node {
  char nodeType; 
  int index;
  double vr; 
  double dr; 
  int child[2];
  bool flag;
};



__global__ void build_circuit(struct node** array, int n, int H, int *num)
{

	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
        unsigned int stride = gridDim.x*blockDim.x;
        unsigned int offset = 0;

	//__shared__ float cache[256];

	int start = 0, end = 0, diff = 0;

	//array of length n with nodes in level order
	struct node* parent = (struct node*)malloc(sizeof(struct node));
        for(int h=0;h<H;h++){
		start = h == 0 ? 0 : h*num[h-1];
		end = start + num[h];
		diff = end - start;
		while(index + offset < diff) {
        	//1 thread should work on multiple nodes
		parent = array[start+index+offset];

		/*Assign dr values depending on parent node*/
		if (parent->nodeType == '+') {
			array[parent->child[0]]->dr = parent->dr;
			array[parent->child[1]]->dr = parent->dr;
		}
		else if (parent->nodeType == '*') {
		/*if bit flag is down, and parent is non-zero, dr(c) = dr(p)*vr(p)/vr(c)*/
			if (parent->dr == 0) {
			//if (cache[start+index+offset] == 0) {
				/*Set all child nodes dr to zero*/
				array[parent->child[0]]->dr = 0;
				array[parent->child[1]]->dr = 0;
			}
			else if (parent->flag) {
				/*Check value of all child nodes*/
				/*if flag is up and child is zero, then dr(c) = dr(p) * vr(p)*/
				if (array[parent->child[0]]->vr == 0) {
					array[parent->child[0]]->dr = parent->dr * parent->vr;
					/*Set all other children dr to zero*/
					array[parent->child[1]]->dr = 0;
				}
        			else {
					array[parent->child[1]]->dr = 0;
					array[parent->child[0]]->dr = parent->dr *
                        		(parent->vr / array[parent->child[0]]->vr);
				}
			}
			else {
				array[parent->child[1]]->dr = parent->dr *
                		(parent->vr / array[parent->child[1]]->vr);
				array[parent->child[0]]->dr = parent->dr *
                		(parent->vr / array[parent->child[0]]->vr);
			}
		}
		//testing code
		//cache[parent->child[0]] = array[parent->child[0]]->dr;
		//cache[parent->child[1]] = array[parent->child[1]]->dr;
		offset += stride;
	}
	__syncthreads();
}
}
